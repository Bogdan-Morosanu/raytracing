
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

namespace rt {
  namespace internal {

    void check_cuda_error(hipError_t result, const char *const val,
			  const char *const file, int line)
    {
      if (result) {
	std::fprintf(stderr, "CUDA error: %d (%s) at %s:%d\n", (int)result, val, file, line);
	hipDeviceReset();
	std::exit(EXIT_FAILURE);
      }
    }
  }
}
